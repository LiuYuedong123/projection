#include "hip/hip_runtime.h"
#include"DataStruct.h"
#include"Line.h"
#include"Point.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#define PI 3.141592653
__global__ void meshLinePosition(Line* rowLine, float sizePixel, float center, float numPixel, float a, float b)
{  //  ����ͼ�����ر߽��������

	for (size_t i = 0; i < numPixel + 1; i++)
	{
		float c = i * sizePixel - center;
		Line tmp(a, b, c);
		rowLine[i] = tmp;
	}
}
__global__ void IntersectPointArray(const Line& Lsd, const Line* meshLine, vector<Point>& pointArray, const ImgParams& imgPar)
{	// ���������������ߵĽ��㣬
	float imgCtrX = imgPar.centerX;
	float imgCtrY = imgPar.centerY;
	float imgLenX = imgPar.sizeX * imgPar.pixelSizeX;
	float imgLenY = imgPar.sizeY * imgPar.pixelSizeY;

	// ��������ͼ���ÿ�еĽ�������ꣻ�ڱ߽�ʱ������ֻ�벿�������ཻ
	auto border = [](float point, float ctr, float len) {return (point <= len - ctr && point >= -ctr); };
	for (size_t i = 0; i < meshLine.size(); i++)
	{
		Point tmp = Line::PointOfIntersection(Lsd, meshLine[i]);
		size_t count = 0;
		if (border(tmp.X(), imgCtrX, imgLenX) && border(tmp.Y(), imgCtrY, imgLenY))
		{
			pointArray.push_back(tmp);
		}

	}
}
__global__ void InterPointMesh(const Line& lsd, Line* rowMesh, Line* colMesh, ImgParams* imgPar, Point* interPoint)
{   //  ������������񽻵㣬�������������������
	vector<Point> rowPoint;  //  ���еĽ���
	vector<Point> colPoint;  //  ���еĽ���
	//������������������ߵĽ���
	IntersectPointArray(lsd, rowMesh, rowPoint, imgPar);
	IntersectPointArray(lsd, colMesh, colPoint, imgPar);
	//  �����������鰴���� �������У��������ߵļнǣ�
	if (lsd.Theta() >= 0 && lsd.Theta() <= PI / 2) // �н�С��90
	{
		sortMergeArray(rowPoint, colPoint, interPoint);
	}
	else  //  �нǴ���90  
	{
		vector<Point> colPoint2;//  col �ĵ���  ���нǴ���90ʱ��X��Y�ɷ���
		for (size_t i = 0; i < colPoint.size(); i++)
		{
			colPoint2.push_back(colPoint[colPoint.size() - 1 - i]);
		}
		sortMergeArray(rowPoint, colPoint2, interPoint);
	}
}
__global__ void backwardProjector(ImgParams* imgPar, float* d_detLine, float* d_proj,float* d_rowLine,
	float* d_colLine)
{





}
extern void ProjBackwardLD(ImgParams* imgPar, DetParams* detPar, float* proj, float* img)
{
	float SOD = detPar->SOD;
	float SDD = detPar->SDD;
	float ctrDet = detPar->centerU; // ʵ��λ�ã�mm��
	float unitSize = detPar->detColSize;
	size_t detNum = detPar->detCol;
	float detLen = detNum * unitSize;
	float projNum = detPar->projNum;
	size_t imgNumX = imgPar->sizeX;
	size_t imgNumY = imgPar->sizeY;
	float pixelSizeX = imgPar->pixelSizeX;
	float pixelSizeY = imgPar->pixelSizeY;
	float imgCtrX = imgPar->centerX;//  ʵ��λ�ã�mm��
	float imgCtrY = imgPar->centerY;

	Point* detCtrPos=new Point[projNum];   //  ̽�����е�����
	float* angleVec=new float[projNum];    //  ͶӰ�Ƕ�
	Point* sourPos = new Point[projNum];     //  Դ����
	for (size_t i = 0; i < projNum; i++)
	{
		float theta = (i * 2 * PI) / projNum;
		angleVec[i] = theta;
		Point tmp((SDD - SOD) * cos(PI + theta), (SDD - SOD) * sin(PI + theta));
		detCtrPos[i] = tmp;
		Point tmp1(SOD * cos(theta), SOD * sin(theta));
		sourPos[i] = tmp1;
	}
	float* detPosRel = new float[detNum];//  ����̽�����ĵ�Ԫ����е������,һά
	for (size_t i = 0; i < detNum; i++)
	{
		float temp = (0.5 + i) * unitSize - ctrDet;
		detPosRel[i] = temp;
	}
	Line* rowLine = new Line[pixelSizeY + 1];//  ��������������
	Line* colLine = new Line[pixelSizeX + 1];
	meshLinePosition(rowLine, pixelSizeY, imgCtrY, imgNumY, 0, -1);
	meshLinePosition(colLine, pixelSizeX, imgCtrX, imgNumX, -1, 0);
	Line* detLine = new Line[projNum * detNum];
	for (size_t p = 0; p < projNum; p++)  //  Ԥ�Ȱ��õ������ݴ�������������һ��Ҫ������
	{
		for (size_t i = 0; i < detNum; i++)
		{
			Point Angle(sin(angleVec[p]), -cos(angleVec[p]));
			Point unitPos = detCtrPos[p] + Angle * detPosRel[i];
			Line temp(unitPos, sourPos[p]);
			detLine[i + p * detNum] = temp;
		}
	}
	delete[] detCtrPos;
	delete[] angleVec;
	delete[] sourPos;
	delete[] detPosRel;
	//  ��������ͶӰ



	delete[] rowLine;
	delete[] colLine;
	delete[] detLine;
}