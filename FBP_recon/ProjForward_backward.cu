#include "hip/hip_runtime.h"
#include"DataStruct.h"
#include"Line.h"
#include"Point.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#define PI 3.141592653
struct position //�������ص�Ķ�ά����
{
	size_t x;
	size_t y;
};
void sortMergeArray(const vector<Point>& a1, const vector<Point>& a2, vector<Point>& merge)
{// �������Ѿ��ź�����������鰴�������ų�һ������

	size_t i = 0, j = 0;
	while (i < a1.size() && j < a2.size())
	{
		if (a1[i].Y() < a2[j].Y())
		{
			merge.push_back(a1[i]); ++i;
		}
		else {
			merge.push_back(a2[j]); ++j;
		}
	}
	if (i < a1.size())
	{
		for (size_t s = i; s < a1.size(); s++)
		{
			merge.push_back(a1[i++]);
		}
	}
	if (j < a2.size())
	{
		for (size_t s = j; s < a2.size(); s++)
		{
			merge.push_back(a2[j++]);
		}
	}
}
void pixelIndex2D(const vector<Point>& interPoint, vector<position>& pixelIndex, const ImgParams& imgPar)
{  // �������ߴ��������صĶ�ά����
	float imgCtrX = imgPar.centerX;
	float imgCtrY = imgPar.centerY;
	float psizeX = imgPar.pixelSizeX;
	float psizeY = imgPar.pixelSizeY;
	position tmp;
	for (size_t i = 0; i < interPoint.size() - 1; i++)
	{
		Point mid = Point::MidPoint(interPoint[i], interPoint[i + 1]);
		tmp.x = ((mid.X() + imgCtrX) / psizeX);
		tmp.y = ((mid.Y() + imgCtrY) / psizeY);
		pixelIndex.push_back(tmp);
	}
}
__global__ void meshLinePosition(Line* rowLine, float sizePixel, float center, float numPixel, float a, float b)
{  //  ����ͼ�����ر߽��������

	for (size_t i = 0; i < numPixel + 1; i++)
	{
		float c = i * sizePixel - center;
		Line tmp(a, b, c);
		rowLine[i] = tmp;
	}
}
__global__ void IntersectPointArray(Line Lsd, Line* meshLine,size_t mesh_size, Point* pointArray,ImgParams*imgPar)
{	// ���������������ߵĽ��㣬
	float imgCtrX = imgPar->centerX;
	float imgCtrY = imgPar->centerY;
	float imgLenX = imgPar->sizeX * imgPar->pixelSizeX;
	float imgLenY = imgPar->sizeY * imgPar->pixelSizeY;

	// ��������ͼ���ÿ�еĽ�������ꣻ�ڱ߽�ʱ������ֻ�벿�������ཻ
	auto border = [](float point, float ctr, float len) {return (point <= len - ctr && point >= -ctr); };
	for (size_t i = 0; i < mesh_size; i++)
	{
		Point tmp = Line::PointOfIntersection(Lsd, meshLine[i]);
		size_t count = 0;
		if (border(tmp.X(), imgCtrX, imgLenX) && border(tmp.Y(), imgCtrY, imgLenY))
		{
			pointArray.push_back(tmp);
		}

	}
}
__global__ void InterPointMesh(const Line& lsd, Line* rowMesh, Line* colMesh, ImgParams* imgPar, Point* interPoint)
{   //  ������������񽻵㣬�������������������
	vector<Point> rowPoint;  //  ���еĽ���
	vector<Point> colPoint;  //  ���еĽ���
	//������������������ߵĽ���
	IntersectPointArray(lsd, rowMesh, rowPoint, imgPar);
	IntersectPointArray(lsd, colMesh, colPoint, imgPar);
	//  �����������鰴���� �������У��������ߵļнǣ�
	if (lsd.Theta() >= 0 && lsd.Theta() <= PI / 2) // �н�С��90
	{
		sortMergeArray(rowPoint, colPoint, interPoint);
	}
	else  //  �нǴ���90  
	{
		vector<Point> colPoint2;//  col �ĵ���  ���нǴ���90ʱ��X��Y�ɷ���
		for (size_t i = 0; i < colPoint.size(); i++)
		{
			colPoint2.push_back(colPoint[colPoint.size() - 1 - i]);
		}
		sortMergeArray(rowPoint, colPoint2, interPoint);
	}
}
__global__ void backwardProjector(ImgParams* imgPar,DetParams*detPar,float*d_image, Line* d_detLine, float* d_proj,
	Line* d_rowLine,Line* d_colLine)
{	
	size_t detNum = detPar->detCol;
	size_t projNum = detPar->projNum;

	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	size_t tid = idx + idy * blockDim.x * gridDim.x;
	while (tid< detNum* projNum)  
	{
		Point* pointArray;
		// ����ÿ��������ÿ�л�ÿ�����صĽ���
		InterPointMesh(d_detLine[tid], d_rowLine, d_colLine, imgPar, pointArray);
		if (true)
		{
			// �����ཻ�����ؿ�
			pixelIndex2D(pointArray, index2D, imgPar);

		}

		tid += blockDim.x * gridDim.x * blockDim.y * gridDim.y;
	}



}
extern void ProjBackwardLD(ImgParams* imgPar, DetParams* detPar, float* proj, float* img)
{
	float SOD = detPar->SOD;
	float SDD = detPar->SDD;
	float ctrDet = detPar->centerU; // ʵ��λ�ã�mm��
	float unitSize = detPar->detColSize;
	size_t detNum = detPar->detCol;
	float detLen = detNum * unitSize;
	size_t projNum = detPar->projNum;
	size_t imgNumX = imgPar->sizeX;
	size_t imgNumY = imgPar->sizeY;
	float pixelSizeX = imgPar->pixelSizeX;
	float pixelSizeY = imgPar->pixelSizeY;
	float imgCtrX = imgPar->centerX;//  ʵ��λ�ã�mm��
	float imgCtrY = imgPar->centerY;

	Point* detCtrPos=new Point[projNum];   //  ̽�����е�����
	float* angleVec=new float[projNum];    //  ͶӰ�Ƕ�
	Point* sourPos = new Point[projNum];     //  Դ����
	for (size_t i = 0; i < projNum; i++)
	{
		float theta = (i * 2 * PI) / projNum;
		angleVec[i] = theta;
		Point tmp((SDD - SOD) * cos(PI + theta), (SDD - SOD) * sin(PI + theta));
		detCtrPos[i] = tmp;
		Point tmp1(SOD * cos(theta), SOD * sin(theta));
		sourPos[i] = tmp1;
	}
	float* detPosRel = new float[detNum];//  ����̽�����ĵ�Ԫ����е������,һά
	for (size_t i = 0; i < detNum; i++)
	{
		float temp = (0.5 + i) * unitSize - ctrDet;
		detPosRel[i] = temp;
	}
	Line* rowLine = new Line[pixelSizeY + 1];//  ��������������
	Line* colLine = new Line[pixelSizeX + 1];
	meshLinePosition(rowLine, pixelSizeY, imgCtrY, imgNumY, 0, -1);
	meshLinePosition(colLine, pixelSizeX, imgCtrX, imgNumX, -1, 0);
	Line* detLine = new Line[projNum * detNum];
	for (size_t p = 0; p < projNum; p++)  //  Ԥ�Ȱ��õ������ݴ�������������һ��Ҫ������
	{
		for (size_t i = 0; i < detNum; i++)
		{
			Point Angle(sin(angleVec[p]), -cos(angleVec[p]));
			Point unitPos = detCtrPos[p] + Angle * detPosRel[i];
			Line temp(unitPos, sourPos[p]);
			detLine[i + p * detNum] = temp;
		}
	}   /// ��һ���ӵ��豸������
	delete[] detCtrPos;
	delete[] angleVec;
	delete[] sourPos;
	delete[] detPosRel;
	//  ��������ͶӰ
	float* d_img;
	float* d_proj;
	Line* d_deline;
	Line* d_rowLine;
	Line* d_colLine;
	ImgParams* d_imgPar;
	DetParams* d_detPar;
	hipMalloc((void**)&d_proj, sizeof(float) * projNum * detNum);
	hipMalloc((void**)&d_deline, sizeof(Line) * projNum * detNum);// Ϊ���߿����Դ�projNum * detNum
	hipMalloc((void**)&d_img, sizeof(float)* imgNumX* imgNumY);  // Ϊimg �����Դ�
	hipMalloc((void**)&d_imgPar, sizeof(ImgParams));
	hipMalloc((void**)&d_rowLine, sizeof(Line)*(pixelSizeY + 1));
	hipMalloc((void**)&d_colLine, sizeof(Line)*(pixelSizeX + 1));
	hipMalloc((void**)&d_detPar, sizeof(DetParams)); 

	hipMemcpy(d_detPar, detPar, sizeof(ImgParams), hipMemcpyHostToDevice);//  �������ݵ��Դ�
	hipMemcpy(d_imgPar, imgPar, sizeof(DetParams), hipMemcpyHostToDevice);
	hipMemcpy(d_deline, detLine, sizeof(Line) * projNum * detNum, hipMemcpyHostToDevice);
	hipMemcpy(d_proj, proj, sizeof(float) * projNum * detNum, hipMemcpyHostToDevice);
	hipMemcpy(d_img, img, sizeof(float) * imgNumX * imgNumY, hipMemcpyHostToDevice);
	hipMemcpy(d_rowLine, rowLine, sizeof(Line) * (pixelSizeY + 1), hipMemcpyHostToDevice);
	hipMemcpy(d_colLine, colLine, sizeof(Line) * (pixelSizeX + 1), hipMemcpyHostToDevice);

	delete[] rowLine;
	delete[] colLine;
	delete[] detLine;

	dim3 block(32, 32);
	dim3 grid(256, 32);

	backwardProjector << <grid, block >> > (d_imgPar, d_img, d_detPar,d_deline, d_proj, d_rowLine, d_colLine);
	hipMemcpy(img, d_img, sizeof(float) * imgNumX * imgNumY, hipMemcpyDeviceToHost);
	hipFree(d_img);
	hipFree(d_proj);
	hipFree(d_deline);
	hipFree(d_rowLine);
	hipFree(d_colLine);
	hipFree(d_imgPar);
}